#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>
#include <math.h>
#include <iostream>
#include <string>
#include <vector>
#include "utils.hpp"
#include "update.hpp"
#include "memoryManager.hpp"
#include "cuStinger.hpp"
#include <chrono>
#include <iomanip>
#include <cstdlib>
#include <ctime>
#include <algorithm>

using namespace std;

// CUDA error checking macro
#define CHECK_CUDA(call) {                                            \
    hipError_t err = call;                                           \
    if (err != hipSuccess) {                                         \
        fprintf(stderr, "CUDA error at %s:%d: %s\n",                  \
                __FILE__, __LINE__, hipGetErrorString(err));         \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
}

struct Update {
    bool is_addition; // true if edge is added, false if removed
    vertexId_t u, v;  // edge between vertices u and v
};

/**
 * CUDA kernel for BFS traversal
 * @param numVertices Number of vertices in the graph
 * @param d_offsets CSR offsets array (device)
 * @param d_edges CSR edges array (device)
 * @param d_distances Distance array (device)
 * @param d_frontier Current frontier vertices (device)
 * @param frontierSize Size of current frontier
 * @param d_next_frontier Next frontier vertices (device)
 * @param d_next_frontier_size Size of next frontier (device)
 * @param currentLevel Current BFS level
 */


//////////////////////////////////////
// BFS Kernel
//////////////////////////////////////
__global__ void bfs_kernel(const cuStinger::cusVertexData* dVD,
    const vertexId_t nv,
    const int current_level,
    const int* frontier,
    const int frontier_size,
    int* levels,
    vertexId_t* next_frontier,
    int* next_count) {
  // thread id
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < frontier_size) {
    int v = frontier[tid];  // current vertex from frontier
    int numNeighbors = dVD->used[v];
    // reinterpret pointer to edge memory
    vertexId_t* nbrArray = reinterpret_cast<vertexId_t*>(dVD->edMem[v]);
    for (int i = 0; i < numNeighbors; i++) {
      vertexId_t nbr = nbrArray[i];
      // printf("v-->nbr: %d-->%d\n", v,nbr);
      // If not yet discovered then set level and add to next frontier.
      if (atomicCAS(&levels[nbr], -1, current_level + 1) == -1) {
        int pos = atomicAdd(next_count, 1);
        next_frontier[pos] = nbr;
      }
    }
  }
}

//////////////////////////////////////
// Host-Side BFS Implementation
//////////////////////////////////////
int* runBFS(cuStinger* graph, int source ,int* h_levels,int* d_frontier) {
    // nv = number of vertices
    int nv = graph->nv;
    // array of levels on GPU
    int* d_levels;

    // if CUDA has error -> inform
    CHECK_CUDA(hipMalloc((void**)&d_levels, nv * sizeof(int)));

    // copy memory from host to device
    CHECK_CUDA(hipMemcpy(d_levels, h_levels, nv * sizeof(int), hipMemcpyHostToDevice));

    // Allocate frontiers on device
    // whichever is the border
    // int* d_frontier;

    // 0 - source (d_frontier = &source)
    // 1 - vertices associated
    // ...

    // next border
    int* d_next_frontier;
    // CHECK_CUDA(hipMalloc((void**)&d_frontier, nv * sizeof(vertexId_t)));
    CHECK_CUDA(hipMalloc((void**)&d_next_frontier, nv * sizeof(vertexId_t)));

    // Start frontier contains only the source vertex
    // how many elements in CPU queue
    int h_frontier_size = 1;
    CHECK_CUDA(hipMemcpy(d_frontier, &source, sizeof(int), hipMemcpyHostToDevice));

    // Allocate device counter for the next frontier
    int* d_next_count;
    CHECK_CUDA(hipMalloc((void**)&d_next_count, sizeof(int)));

    int current_level = 0;
    while (h_frontier_size > 0) {
        // Reset next frontier count
        CHECK_CUDA(hipMemset(d_next_count, 0, sizeof(int)));

        // Launch BFS kernel
        int blockSize = 256;
        // which thread runs where 
        int gridSize = (h_frontier_size + blockSize - 1) / blockSize;
        bfs_kernel<<<gridSize, blockSize>>>(graph->dVD, nv, current_level, d_frontier,
                                            h_frontier_size, d_levels, d_next_frontier, d_next_count);
        // cout<<"BFS KERNEL LAUNCHED"<<endl;
        CHECK_CUDA(hipDeviceSynchronize());

        // Copy next frontier size back to host
        int h_next_count;
        CHECK_CUDA(hipMemcpy(&h_next_count, d_next_count, sizeof(int), hipMemcpyDeviceToHost));

        // Swap frontiers
        int* temp = d_frontier;
        d_frontier = d_next_frontier;
        d_next_frontier = temp;

        h_frontier_size = h_next_count;
        current_level++;
    }

    // Copy levels array back to host
    CHECK_CUDA(hipMemcpy(h_levels, d_levels, nv * sizeof(int), hipMemcpyDeviceToHost));

    ///////Print BFS levels
    for (int i = 0; i < nv; i++) {
        // printf("Vertex %d: Level %d\n", i, h_levels[i]);
    }

    // Free allocated memory
    // free(h_levels);
    CHECK_CUDA(hipFree(d_levels));
    CHECK_CUDA(hipFree(d_frontier));
    CHECK_CUDA(hipFree(d_next_frontier));
    CHECK_CUDA(hipFree(d_next_count));
    return h_levels;
}

//////////////////////////////////////
// BFS Kernel using Offset and Adjacency Array
//////////////////////////////////////
__global__ void bfs_kernel_offset_adj(const int* offset,
  const int* adjacency,
  const int* frontier,
  const int frontier_size,
  int* levels,
  int* next_frontier,
  int* next_count,
  const int current_level) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;

if (tid < frontier_size) {
int v = frontier[tid];  // Current vertex in the frontier
int start = offset[v];  // Start of neighbors in adjacency list
int end = offset[v + 1]; // End of neighbors in adjacency list

for (int i = start; i < end; i++) {
int nbr = adjacency[i];
if (atomicCAS(&levels[nbr], -1, current_level + 1) == -1) {
int pos = atomicAdd(next_count, 1);
next_frontier[pos] = nbr;
}
}
}
}


//////////////////////////////////////
// Host-Side BFS Implementation
//////////////////////////////////////
int* runBFS_offset_adj(const int* h_offset,
  const int* h_adjacency,
  int nv,
  int source) {
// Host levels array
int* h_levels = (int*)malloc(nv * sizeof(int));
for (int i = 0; i < nv; i++) {
h_levels[i] = -1;  // Initialize levels to -1
}
h_levels[source] = 0;  // Set the source level to 0

// Device pointers
int *d_offset, *d_adjacency, *d_levels, *d_frontier, *d_next_frontier, *d_next_count;

// Allocate device memory
CHECK_CUDA(hipMalloc(&d_offset, (nv + 1) * sizeof(int)));
CHECK_CUDA(hipMalloc(&d_adjacency, h_offset[nv] * sizeof(int)));
CHECK_CUDA(hipMalloc(&d_levels, nv * sizeof(int)));
CHECK_CUDA(hipMalloc(&d_frontier, nv * sizeof(int)));
CHECK_CUDA(hipMalloc(&d_next_frontier, nv * sizeof(int)));
CHECK_CUDA(hipMalloc(&d_next_count, sizeof(int)));

// Copy graph data to device
CHECK_CUDA(hipMemcpy(d_offset, h_offset, (nv + 1) * sizeof(int), hipMemcpyHostToDevice));
CHECK_CUDA(hipMemcpy(d_adjacency, h_adjacency, h_offset[nv] * sizeof(int), hipMemcpyHostToDevice));
CHECK_CUDA(hipMemcpy(d_levels, h_levels, nv * sizeof(int), hipMemcpyHostToDevice));

// Initialize the source vertex in the frontier
int h_frontier_size = 1;
CHECK_CUDA(hipMemcpy(d_frontier, &source, sizeof(int), hipMemcpyHostToDevice));

int current_level = 0;

while (h_frontier_size > 0) {
// Reset next frontier count
CHECK_CUDA(hipMemset(d_next_count, 0, sizeof(int)));

// Launch BFS kernel
int blockSize = 256;
int gridSize = (h_frontier_size + blockSize - 1) / blockSize;
bfs_kernel_offset_adj<<<gridSize, blockSize>>>(d_offset, d_adjacency,
                                  d_frontier, h_frontier_size,
                                  d_levels, d_next_frontier,
                                  d_next_count, current_level);
CHECK_CUDA(hipDeviceSynchronize());

// Copy next frontier size back to host
int h_next_count;
CHECK_CUDA(hipMemcpy(&h_next_count, d_next_count, sizeof(int), hipMemcpyDeviceToHost));

// Swap frontiers
int* temp = d_frontier;
d_frontier = d_next_frontier;
d_next_frontier = temp;

h_frontier_size = h_next_count;
current_level++;
}

// Copy levels back to host
CHECK_CUDA(hipMemcpy(h_levels, d_levels, nv * sizeof(int), hipMemcpyDeviceToHost));

// Free device memory
CHECK_CUDA(hipFree(d_offset));
CHECK_CUDA(hipFree(d_adjacency));
CHECK_CUDA(hipFree(d_levels));
CHECK_CUDA(hipFree(d_frontier));
CHECK_CUDA(hipFree(d_next_frontier));
CHECK_CUDA(hipFree(d_next_count));

return h_levels;
}


//////////////////////////////////////
// BFS Update Kernel for Edge Additions
//////////////////////////////////////
__global__ void bfs_update_add_kernel(const cuStinger::cusVertexData* dVD,  
    const int* add_edges,   // packed as: [u0, v0, u1, v1, ...]  
    const int num_edges,  
    int* levels,  
    vertexId_t* update_frontier,  
    int* update_count) {  
  int tid = blockIdx.x * blockDim.x + threadIdx.x; 
  if (tid < num_edges) {  

    int u = add_edges[2 * tid];  
    int v = add_edges[2 * tid + 1];  
    if (levels[u] != -1) {  
      int new_level = levels[u] + 1;  
      // Use atomicCAS to update levels[v] only if it is still undiscovered (-1)
      int old = atomicCAS(&levels[v], -1, new_level);  
      
      int old_2 = atomicMin(&levels[v], new_level);
      

      if (old == -1 || old_2>new_level) {  // Successful update: v was undiscovered  
        int pos = atomicAdd(update_count, 1);  
        update_frontier[pos] = v;  
      }  
    }  
  }  
}



//////////////////////////////////////
// BFS Update Kernel for Edge Deletions
//////////////////////////////////////
__global__ void bfs_update_del_kernel(const cuStinger::cusVertexData* dVD,
    const int* del_edges,   // packed as: [u0, v0, u1, v1, ...]
    const int num_edges,
    int* levels,
    vertexId_t* update_frontier,
    int* update_count) {
    
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num_edges) {
    int u = del_edges[2 * tid];
    int v = del_edges[2 * tid + 1];
    // Only process if (u,v) was used in the BFS tree:
    // u must have been reached and v’s level must be exactly levels[u] + 1.
    if (levels[u] == -1 || levels[v] != levels[u] + 1)
      return;
      
    // Mark v for recomputation: use atomicExch so that only one thread marks v.
    //Thats wrong as we are marking v as -1 i tmay be connected to other nodes
    int old_level = atomicExch(&levels[v], -1);
    // If v was not already marked, add it to the update frontier.
    if(old_level != -1) {
      int pos = atomicAdd(update_count, 1);
      update_frontier[pos] = v;
      // printf("---------inside kernel update_frontier: %d-----v:%d-----\n", update_frontier[pos],v); 
    }
  }
}



//////////////////////////////////////
// BFS Recompute Kernel for Propagation
//////////////////////////////////////

__global__ void bfs_recompute_kernel(const cuStinger::cusVertexData* dVD,
    const int nv,
    const int* frontier,
    const int frontier_size,
    int* levels,
    vertexId_t* next_frontier,
    int* next_count) {
    
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < frontier_size) {
    int v = frontier[tid];
    // Recompute the new level for v.
    int candidate = INT_MAX;
    int numNeighbors = dVD->used[v];
    vertexId_t* nbrArray = reinterpret_cast<vertexId_t*>(dVD->edMem[v]);
    
    // For each neighbor, consider candidate level = levels[nbr] + 1.
    // Only consider neighbors with a valid (non-INT_MAX, non -1) level.
    for (int i = 0; i < numNeighbors; i++) {
      int nbr = nbrArray[i];
      int nbr_level = levels[nbr];
      if(nbr_level >= 0 && nbr_level != INT_MAX) {
        int old = min(candidate, nbr_level + 1);
      }
    }
    
    // If no valid candidate was found, v remains unreachable.
    int new_level = (candidate == INT_MAX) ? -1 : candidate;
    
    // Update v’s level if it differs from the (marked) value.
    // Note: v was marked as INT_MAX, so we expect new_level != INT_MAX.
    int old_level = INT_MAX;
    old_level = atomicExch(&levels[v], new_level);

    if (new_level != old_level) {
      for (int i = 0; i < numNeighbors; i++) {
        int nbr = nbrArray[i];
        // If neighbor has a valid level and its current level is greater than new_level + 1,
        // try to update it.
        if (levels[nbr] != -1 && levels[nbr] > new_level + 1) {
          int prev = atomicMin(&levels[nbr], new_level + 1);
          if (prev > new_level + 1) {
            int pos = atomicAdd(next_count, 1);
            next_frontier[pos] = nbr;
          }
        }
      }
    }
  }
}

//////////////////////////////////////
// BFS Kernel to handke updates
//////////////////////////////////////
__global__ void bfs_kernel_update(const cuStinger::cusVertexData* dVD,
    const vertexId_t nv,
    // int current_level,
    const int* frontier,
    const int frontier_size,
    int* levels,
    vertexId_t* next_frontier,
    int* next_count
  ) {
  // thread id
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < frontier_size && tid < nv) {
    // printf("entered kernel with tid : %d\n", tid);
    int v = frontier[tid];  // current vertex from frontier
    // printf("v valid: %d\n", v);
    // printf("here\n");
    // printf("dVD->used[v]: %lld\n", dVD->used[v]);
    int numNeighbors = dVD->used[v];
    // printf("numNeighbors: %d\n", numNeighbors);
    vertexId_t* nbrArray = reinterpret_cast<vertexId_t*>(dVD->edMem[v]);
    // printf("entering for loop\n");
    for (int i = 0; i < numNeighbors; i++) {
      // printf("entered for loop\n");
      vertexId_t nbr = nbrArray[i];
      if(nbr == -1) {
        // printf("nbr is -1\n");
        continue;
      }
      // printf("v-->nbr:%d--> %d\n",v, nbr);
      int current_level = levels[v];
      // printf("cuurrent_level[%d]: %d\n",v, current_level);
      // If not yet discovered then set level and add to next frontier.
      // printf("nbr: %d\n", nbr);
      int old = atomicMin(&levels[nbr], current_level + 1);
      // printf("old: %d\n", old);
      if (old == -1 || old > current_level + 1) {
        // printf("entered if condition\n");
        int pos = atomicAdd(next_count, 1);
        next_frontier[pos] = nbr;
        // printf("v: %d--> next_frontier[%d]: %d\n",v,pos, next_frontier[pos]);
      }
    }
    // printf("leaving for loop\n");
  }
}

//////////////////////////////////////
// BFS Kernel betweeen src and v
//////////////////////////////////////
__global__ void bfs_kernel_bt_src_v(const cuStinger::cusVertexData* dVD,
  const vertexId_t nv,
  const int current_level,
  const int* frontier,
  const int frontier_size,
  int* levels,
  vertexId_t* next_frontier,
  int* next_count,
vertexId_t* v_d,
int* found,
int* len_v,
int * found_ver_arr) {
// thread id
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < frontier_size&& frontier_size<nv && found<len_v) {
  int v = frontier[tid];  // current vertex from frontier
  // printf("----v: %d\n", v);
  if (v < 0 || v >= nv) return;
  int numNeighbors = dVD->used[v];
  // reinterpret pointer to edge memory
  vertexId_t* nbrArray = reinterpret_cast<vertexId_t*>(dVD->edMem[v]);

  if (nbrArray == nullptr) return;
  for (int i = 0; i < numNeighbors; i++) {
    vertexId_t nbr = nbrArray[i];
    int already_found = 0;
    // printf("v-->nbr:%d--> %d\n", v,nbr);

    // printf("---v: %d\n", (v_d[1]));
    // printf("---len_v: %d\n", *len_v);

    for(int j=0;j<*len_v;j++) {
      // printf("-v_d[j]: %d\n", v_d[j]);
      if (nbr == v_d[j]) {
      for (int k=0;k<*len_v;k++){
      
        if(nbr == found_ver_arr[k]){
          // printf("nbr already found\n");
          already_found = 1;
        }
      }
      if(!already_found){
      found_ver_arr[*found] = nbr;
      // printf("---found_ver_arr %d\n",found_ver_arr[*found]);
      atomicAdd(found, 1); // Signal discovery}
    }
  }}
    // printf("v-->nbr: %d-->%d\n", v,nbr);
 
    // printf("v-->nbr: %d-->%d\n", v,nbr);
    // If not yet discovered then set level and add to next frontier.
    if (atomicCAS(&levels[nbr], -1, current_level + 1) == -1) {
      int pos = atomicAdd(next_count, 1);
      next_frontier[pos] = nbr;
    }
  }
}
}

//////////////////////////////////////
// Host-Side BFS Implementation
//////////////////////////////////////
int* runBFS_bt_src_v(cuStinger* graph, int source ,int* h_levels,int* d_frontier,vertexId_t* v, int* len_v , int * found_cnt, int * found_ver_arr) {
  // nv = number of vertices
  int nv = graph->nv;
  // array of levels on GPU
  int* d_levels;
  vertexId_t * d_found_ver_arr;

  // if CUDA has error -> inform
  CHECK_CUDA(hipMalloc((void**)&d_levels, nv * sizeof(int)));
  CHECK_CUDA(hipMalloc((void**)&d_found_ver_arr, (*len_v) * sizeof(vertexId_t)));
  // printf("mem allocation success for d levels and d found ver arr\n");
  // copy memory from host to device
  CHECK_CUDA(hipMemcpy(d_levels, h_levels, nv * sizeof(int), hipMemcpyHostToDevice));

  // Allocate frontiers on device
  // whichever is the border
  // int* d_frontier;

  // 0 - source (d_frontier = &source)
  // 1 - vertices associated
  // ...

  // next border
  int* d_next_frontier;
  // CHECK_CUDA(hipMalloc((void**)&d_frontier, nv * sizeof(vertexId_t)));
  CHECK_CUDA(hipMalloc((void**)&d_next_frontier, nv * sizeof(vertexId_t)));
  // printf("mem allocation success for d next frontier\n");

  // Start frontier contains only the source vertex
  // how many elements in CPU queue
  int h_frontier_size = 1;
  CHECK_CUDA(hipMemcpy(d_frontier, &source, sizeof(int), hipMemcpyHostToDevice));

  // Allocate device counter for the next frontier
  int* d_next_count;
  CHECK_CUDA(hipMalloc((void**)&d_next_count, sizeof(int)));
  // printf("mem allocation success for d next count\n");

  vertexId_t* v_d;
  CHECK_CUDA(hipMalloc((void**)&v_d, (*(len_v))*sizeof(vertexId_t)));
  CHECK_CUDA(hipMemcpy(v_d,v, (*(len_v))*sizeof(vertexId_t),hipMemcpyHostToDevice));
  // printf("mem allocation success for v_d\n");
  // printf("v_d[0]: %d\n", v[1]);

  int* found;
  CHECK_CUDA(hipMalloc((void**)&found, sizeof(int)));
  // printf("mem allocation success for found\n");
  // CHECK_CUDA(hipMemcpy(v_d,v, sizeof(int),hipMemcpyHostToDevice));
  int found_h = 0;
  int* len_v_d;
  CHECK_CUDA(hipMalloc((void**)&len_v_d, sizeof(int)));
  // printf("mem allocation success for len_v_d\n");
  CHECK_CUDA(hipMemcpy(len_v_d,len_v,sizeof(int),hipMemcpyHostToDevice));
  // printf("memcpy success for len v d\n");
  CHECK_CUDA(hipMemcpy(found,&found_h, sizeof(int),hipMemcpyHostToDevice));
  // printf("memcpy success for found\n");
  int current_level = 0;
  // printf("runnig kernel while loop\n");
  while (h_frontier_size > 0&& found_h<*len_v) {
      // Reset next frontier count
      CHECK_CUDA(hipMemset(d_next_count, 0, sizeof(int)));

      // Launch BFS kernel
      int blockSize = 256;
      // which thread runs where 
      int gridSize = (h_frontier_size + blockSize - 1) / blockSize;
      bfs_kernel_bt_src_v<<<gridSize, blockSize>>>(graph->dVD, nv, current_level, d_frontier,
                                          h_frontier_size, d_levels, d_next_frontier, d_next_count, v_d, found,len_v_d,d_found_ver_arr);
      // cout<<"BFS KERNEL LAUNCHED"<<endl;
      CHECK_CUDA(hipDeviceSynchronize());

      // Copy next frontier size back to host
      int h_next_count;
      CHECK_CUDA(hipMemcpy(&h_next_count, d_next_count, sizeof(int), hipMemcpyDeviceToHost));
      // cout<<"h_next_count: "<<h_next_count<<endl;
      // Swap frontiers
      int* temp = d_frontier;
      d_frontier = d_next_frontier;
      d_next_frontier = temp;
      CHECK_CUDA(hipMemcpy(&found_h,found, sizeof(int),hipMemcpyDeviceToHost));
      // cout<<"found_h: "<<found_h<<endl;
      h_frontier_size = h_next_count;
      current_level++;
      // printf("--current_level: %d\n", current_level);
      // printf("--found: %d\n", found_h);
  }

  // Copy levels array back to host
  CHECK_CUDA(hipMemcpy(h_levels, d_levels, nv * sizeof(int), hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(found_ver_arr,d_found_ver_arr, (*len_v) * sizeof(vertexId_t), hipMemcpyDeviceToHost));
  ///////Print BFS levels
  // for (int i = 0; i < nv; i++) {
  //     printf("src - v Vertex %d: Level %d\n", i, h_levels[i]);
  // }
  *found_cnt = found_h;

  // Free allocated memory
  // free(h_levels);
  CHECK_CUDA(hipFree(d_levels));
  CHECK_CUDA(hipFree(d_frontier));
  CHECK_CUDA(hipFree(d_next_frontier));
  CHECK_CUDA(hipFree(d_next_count));
  return h_levels;
}

//////////////////////////////////////
// Host-Side Streaming Update Handler
//////////////////////////////////////
// This function applies a batch of update edges (additions or deletions)
// and then “propagates” the changes via BFS until no further level changes occur.
void updateBFSUpdates(cuStinger* graph, int* h_levels, int* d_frontier,
                      int* h_update_edges, int num_updates, bool isAddition, int source) {
  int nv = graph->nv;

  // Copy update edges to device memory.
  int* d_update_edges;
  CHECK_CUDA(hipMalloc((void**)&d_update_edges, 2 * num_updates * sizeof(int)));
  CHECK_CUDA(hipMemcpy(d_update_edges, h_update_edges,
                        2 * num_updates * sizeof(int), hipMemcpyHostToDevice));

  int* d_levels;
  CHECK_CUDA(hipMalloc((void**)&d_levels, nv * sizeof(int)));

  CHECK_CUDA(hipMemcpy(d_levels, h_levels,
                        nv * sizeof(int), hipMemcpyHostToDevice));

  // Allocate temporary frontier and counter for propagating update changes.
  vertexId_t* d_update_frontier;
  vertexId_t* d_next_frontier;

  vertexId_t* h_update_frontier = (vertexId_t*)malloc(nv * sizeof(vertexId_t));
  CHECK_CUDA(hipMalloc((void**)&d_update_frontier, nv * sizeof(vertexId_t)));
  CHECK_CUDA(hipMalloc((void**)&d_next_frontier, nv * sizeof(vertexId_t)));

  int* d_update_count;
  CHECK_CUDA(hipMalloc((void**)&d_update_count, sizeof(int)));

  int h_update_count = 0;
  int blockSize = 1024;
  int gridSize = (nv + blockSize - 1) / blockSize;

  // Launch the appropriate update kernel.
  if (isAddition) {
    bfs_update_add_kernel<<<gridSize, blockSize>>>(graph->dVD, d_update_edges, num_updates,
                                                     d_levels, d_update_frontier, d_update_count);

      // Get the count of vertices in the update frontier.
    CHECK_CUDA(hipMemcpy(&h_update_count, d_update_count, sizeof(int), hipMemcpyDeviceToHost));
    
    

  // Propagate the update changes until the frontier is empty.
    // int current_level = 0;  // You might want to adjust this if levels are relative to a root.
    // printf("h_update_count: %d\n", h_update_count);
    while (h_update_count > 0) {

        // Reset the counter for the next propagation step.
        CHECK_CUDA(hipMemset(d_update_count, 0, sizeof(int)));
        gridSize = (h_update_count + blockSize - 1) / blockSize;
        // Use the original BFS kernel to propagate updated levels.
        bfs_kernel_update<<<gridSize, blockSize>>>(graph->dVD, nv, d_update_frontier,
                                            h_update_count, d_levels, d_next_frontier, d_update_count);
        CHECK_CUDA(hipDeviceSynchronize());
        // Swap frontiers.
        vertexId_t* temp;
        CHECK_CUDA(hipMalloc((void**)&temp, nv * sizeof(vertexId_t)));
        d_update_frontier = d_next_frontier;
        d_next_frontier = temp;
        // Get the new update frontier count.
        CHECK_CUDA(hipMemcpy(&h_update_count, d_update_count, sizeof(int), hipMemcpyDeviceToHost));
        // printf("h_update_count: %d\n", h_update_count);

        // current_level++;
  }      // Copy levels array back to host
   CHECK_CUDA(hipMemcpy(h_levels, d_levels, nv * sizeof(int), hipMemcpyDeviceToHost));
   // Print BFS levels
   for (int i = 0; i < nv; i++) {
    // printf("Vertex %d: Level %d\n", i, h_levels[i]);
}                                           
           
  } else {
    printf("running delete kernel\n");
    // printf("reached else condition\n");
      
    int del_current_level = 0;
    vertexId_t* v_arr=(vertexId_t*)malloc(num_updates*sizeof(vertexId_t));
    printf("num_updates: %d\n", num_updates);
    for (int i=0;i<num_updates;i++) {
      v_arr[i] = h_update_edges[2*i+1];
      // cout<<"v_arr[i]: "<<v_arr[i]<<endl;
    }
    // printf("starting bfs bt src and v\n");
    int* h_levels_del = (int*)malloc(nv * sizeof(int));
        for (int i = 0; i < nv; i++) {
            h_levels_del[i] = -1;  // -1 indicates undiscovered
        }
        h_levels_del[1] = 0;
    int found = 0;
    vertexId_t * found_ver_arr= (vertexId_t *)malloc(num_updates*sizeof(vertexId_t));
    // printf("---num_updates: %d\n", num_updates);
    // printf("running bfs between src n v\n");
    int* level =runBFS_bt_src_v(graph, source,h_levels_del, d_frontier, v_arr, &num_updates,&found,found_ver_arr);
    // printf("bfs bt src and v done\n");
    // cout<<"found: "<<found<<endl;
    // cout<<"num_updates: "<<num_updates<<endl;
    // bfs_update_del_kernel<<<gridSize, blockSize>>>(graph->dVD, d_update_edges, num_updates,
    //       d_levels, d_update_frontier, d_update_count);
  //  printf("run completed\n");
    if(found<num_updates){
      for (int i=0;i<nv;i++) {
        h_levels[i] = level[i];
      // printf("...Vertex %d: Level %d\n", i, level[i]);

      }
    return;
    }

    for (int i=0;i<=v_arr[num_updates-1];i++) {
      h_levels[i] = level[i];
    }


    h_update_count = found;
    CHECK_CUDA(hipDeviceSynchronize());
    printf("---found: %d\n", found);
    // CHECK_CUDA(hipMemcpy(&num_updates, d_update_count, sizeof(int), hipMemcpyHostToDevice));
    for (int i=0;i<num_updates;i++) {
        // printf("found_ver_arr[%d]: %d\n",i, found_ver_arr[i]);
    }
    
    CHECK_CUDA(hipMemcpy(d_update_frontier,found_ver_arr, num_updates*sizeof(vertexId_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_levels, h_levels, nv * sizeof(int), hipMemcpyHostToDevice));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    // std::cout << "Max Threads Per Block: " << deviceProp.maxThreadsPerBlock << std::endl;
    // std::cout << "Max Grid Size: " << deviceProp.maxGridSize[0] << std::endl;
    assert(d_update_frontier != nullptr);
    assert(&h_update_count != nullptr);
    assert(d_levels != nullptr);
    assert(d_next_frontier != nullptr);
    assert(d_update_count != nullptr);

    // std::cout << "All pointers are valid!" << std::endl;
    while (h_update_count > 0) {

      // Reset the counter for the next propagation step.
      CHECK_CUDA(hipMemset(d_update_count, 0, sizeof(int)));
      gridSize = (h_update_count + blockSize - 1) / blockSize;
      // Use the original BFS kernel to propagate updated levels.
      // printf("running bfs kernel update\n");
      // printf("working till here\n");
      bfs_kernel_update<<<gridSize, blockSize>>>(graph->dVD, nv, d_update_frontier,
                                          h_update_count, d_levels, d_next_frontier, d_update_count);
      // printf("bfs kernel update done\n");
      // printf("h_update_count: %d\n", h_update_count);
      hipError_t err = hipGetLastError();
if (err != hipSuccess) {
    printf("CUDA Kernel Error: %s\n", hipGetErrorString(err));
}
      CHECK_CUDA(hipDeviceSynchronize());
      // Swap frontiers.
      vertexId_t* temp;
      CHECK_CUDA(hipMalloc((void**)&temp, nv * sizeof(vertexId_t)));
      d_update_frontier = d_next_frontier;
      d_next_frontier = temp;
      CHECK_CUDA(hipFree(temp));
      // Get the new update frontier count.
      CHECK_CUDA(hipMemcpy(&h_update_count, d_update_count, sizeof(int), hipMemcpyDeviceToHost));

}      // Copy levels array back to host
 CHECK_CUDA(hipMemcpy(h_levels, d_levels, nv * sizeof(int), hipMemcpyDeviceToHost));
 // Print BFS levels
 for (int i = 0; i < nv; i++) {
  // printf("----Vertex %d: Level %d\n", i, h_levels[i]);
}  






  
  }                       

  CHECK_CUDA(hipDeviceSynchronize());


   

  // Free temporary device memory.
  if (isAddition) {
    CHECK_CUDA(hipFree(d_update_frontier));
  } else {
    // CHECK_CUDA(hipFree(d_next_frontier));
  }
  // CHECK_CUDA(hipFree(d_update_edges));
  CHECK_CUDA(hipFree(d_levels));
  if (!isAddition ) {
    CHECK_CUDA(hipFree(d_update_frontier));
  }
  // CHECK_CUDA(hipFree(d_update_frontier));
  CHECK_CUDA(hipFree(d_update_count));
}


// Printer utility function for cuStinger
void printcuStingerUtility(cuStinger custing, bool allInfo) {
    length_t used, allocated;
    used = custing.getNumberEdgesUsed();
    allocated = custing.getNumberEdgesAllocated();
    if (allInfo)
        cout << "," << used << "," << allocated << "," << (float)used / (float)allocated;  
    else
        cout << "," << (float)used / (float)allocated;
}

// Generate random edge updates
void generateEdgeUpdates(length_t nv, length_t numEdges, vertexId_t* edgeSrc, vertexId_t* edgeDst, int* update_edges) {
    for (int e = 0; e < numEdges; e++) {
        edgeSrc[e] = update_edges[2*e] % nv;
        edgeDst[e] = update_edges[2*e+1] % nv;
    }
}

// RMAT edge generation (helper functions)
typedef struct dxor128_env {
    unsigned x, y, z, w;
} dxor128_env_t;

double dxor128(dxor128_env_t * e) {
    unsigned t = e->x ^ (e->x << 11);
    e->x = e->y; e->y = e->z; e->z = e->w;
    e->w = (e->w ^ (e->w >> 19)) ^ (t ^ (t >> 8));
    return e->w * (1.0 / 4294967296.0);
}

void dxor128_init(dxor128_env_t * e) {
    e->x = 123456789;
    e->y = 362436069;
    e->z = 521288629;
    e->w = 88675123;
}

void dxor128_seed(dxor128_env_t * e, unsigned seed) {
    e->x = 123456789;
    e->y = 362436069;
    e->z = 521288629;
    e->w = seed;
}

void rmat_edge(int64_t * iout, int64_t * jout, int SCALE, double A, double B, double C, double D, dxor128_env_t * env)
{
    int64_t i = 0, j = 0;
    int64_t bit = ((int64_t) 1) << (SCALE - 1);

    while (1) {
        const double r = ((double) rand() / (RAND_MAX));
        if (r > A) {
            if (r <= A + B)
                j |= bit;
            else if (r <= A + B + C)
                i |= bit;
            else {
                j |= bit;
                i |= bit;
            }
        }
        if (1 == bit)
            break;

        A *= (9.5 + ((double) rand() / (RAND_MAX))) / 10;
        B *= (9.5 + ((double) rand() / (RAND_MAX))) / 10;
        C *= (9.5 + ((double) rand() / (RAND_MAX))) / 10;
        D *= (9.5 + ((double) rand() / (RAND_MAX))) / 10;

        {
            const double norm = 1.0 / (A + B + C + D);
            A *= norm; B *= norm; C *= norm;
        }
        D = 1.0 - (A + B + C);
        bit >>= 1;
    }
    *iout = i;
    *jout = j;
}

// Generate RMAT edge updates
void generateEdgeUpdatesRMAT(length_t nv, length_t numEdges, vertexId_t* edgeSrc, vertexId_t* edgeDst,
                             double A, double B, double C, double D, dxor128_env_t * env)
{
    int64_t src, dst;
    int scale = (int)log2(double(nv));
    for (int32_t e = 0; e < numEdges; e++) {
        rmat_edge(&src, &dst, scale, A, B, C, D, env);
        edgeSrc[e] = src;
        edgeDst[e] = dst;
    }
}

int* gen_rand_edges(int size, int nv) {
  std::srand(std::time(0)); // Seed for random number generation
  int* result = new int[size * 2];

  for (int i = 0; i < size; ++i) {
      int u = std::rand() % nv + 1; // Generate a random vertex u
      int v = std::rand() % nv + 1; // Generate a random vertex v

      // Ensure u < v
      while (u == v) {
          v = std::rand() % nv + 1;
      }
      if (u > v) std::swap(u, v);

      // Add u and v to the result
      result[2 * i] = u;
      result[2 * i + 1] = v;
  }


  return result;
}

int main(const int argc, char *argv[])
{  

    int device = 0;
    hipSetDevice(device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
 
    length_t nv, ne, *off;
    vertexId_t *adj;
    int isRmat = 0;

    if (argc < 3) {
        cout << "Usage: " << argv[0] << " <graph_file> <graphName> [options]\n";
        return 1;
    }
 
    char* graphName = argv[2];
    srand(100);

    bool isDimacs, isSNAP, isMM;
    string filename(argv[1]);
    isDimacs = (filename.find(".graph") != string::npos);
    isSNAP   = (filename.find(".txt") != string::npos);
    isMM     = (filename.find(".mtx") != string::npos);
    isRmat   = (filename.find("kron") != string::npos);

    bool undirected = hasOption("--undirected", argc, argv);

    if (isDimacs) {
        readGraphDIMACS(argv[1], &off, &adj, &nv, &ne, isRmat);
    } else if (isSNAP) {
        readGraphSNAP(argv[1], &off, &adj, &nv, &ne, undirected);
    } else if (isMM) {
        readGraphMatrixMarket(argv[1], &off, &adj, &nv, &ne, undirected);
    } else { 
        cout << "Unknown graph type" << endl;
        return 1;
    }
 

    // Set up for cuStinger
    hipEvent_t ce_start, ce_stop;
    cuStingerInitConfig hipInit;
    hipInit.initState   = eInitStateCSR;
    hipInit.maxNV       = nv + 1;
    hipInit.useVWeight  = false;
    hipInit.isSemantic  = false;
    hipInit.useEWeight  = false;
    
    hipInit.csrNV       = nv;
    hipInit.csrNE       = ne;
    hipInit.csrOff      = off;
    hipInit.csrAdj      = adj;
    hipInit.csrVW       = NULL;
    hipInit.csrEW       = NULL;
    cuStinger custing2(defaultInitAllocater, defaultUpdateAllocater);
    //Initialize cuStinger DataStructure
    custing2.initializeCuStinger(hipInit);

    
    int* h_levels = (int*)malloc(nv * sizeof(int));
    for (int i = 0; i < nv; i++) {
        h_levels[i] = -1;  // -1 indicates undiscovered
    }
    int source = 1;
    h_levels[source] = 0;
    int* d_frontier;
    CHECK_CUDA(hipMalloc((void**)&d_frontier, nv * sizeof(vertexId_t)));

    cout << "Running BFS on the input graph ...\n";
    auto start = std::chrono::high_resolution_clock::now();

    int *levels = runBFS(&custing2, source, h_levels, d_frontier);

    auto end = std::chrono::high_resolution_clock::now();

    std::cout << "Finished BFS.\n\n";
    std::cout << "nv: " << hipInit.csrNV << " ne: " << hipInit.csrNE << std::endl;

    // Calculate total duration in nanoseconds
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);

    // Extract each time unit
    auto hours = std::chrono::duration_cast<std::chrono::hours>(duration);
    auto minutes = std::chrono::duration_cast<std::chrono::minutes>(duration % std::chrono::hours(1));
    auto seconds = std::chrono::duration_cast<std::chrono::seconds>(duration % std::chrono::minutes(1));
    auto milliseconds = std::chrono::duration_cast<std::chrono::milliseconds>(duration % std::chrono::seconds(1));
    auto microseconds = std::chrono::duration_cast<std::chrono::microseconds>(duration % std::chrono::milliseconds(1));
    auto nanoseconds = std::chrono::duration_cast<std::chrono::nanoseconds>(duration % std::chrono::microseconds(1));

    std::cout << "Time taken for BFS: "
              << hours.count() << " hours, "
              << minutes.count() << " minutes, "
              << seconds.count() << " seconds, "
              << milliseconds.count() << " milliseconds, "
              << microseconds.count() << " microseconds, "
              << nanoseconds.count() << " nanoseconds" << std::endl;



              printf("Running BFS with edge additions...\n");


              /// bfs using offset and adjacency list


         start = std::chrono::high_resolution_clock::now();
        runBFS_offset_adj(off, adj, nv,source);
        end = std::chrono::high_resolution_clock::now();
        
        // Calculate total duration
        duration =  std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
        
         hours = std::chrono::duration_cast<std::chrono::hours>(duration);
       minutes = std::chrono::duration_cast<std::chrono::minutes>(duration % std::chrono::hours(1));
      seconds = std::chrono::duration_cast<std::chrono::seconds>(duration % std::chrono::minutes(1));
         milliseconds = std::chrono::duration_cast<std::chrono::milliseconds>(duration % std::chrono::seconds(1));
        microseconds = std::chrono::duration_cast<std::chrono::microseconds>(duration % std::chrono::milliseconds(1));
        nanoseconds = std::chrono::duration_cast<std::chrono::nanoseconds>(duration % std::chrono::microseconds(1));

        std::cout << "Time taken for bfs using off and adj : "
                  << hours.count() << " hours, "
                  << minutes.count() << " minutes, "
                  << seconds.count() << " seconds, "
                  << milliseconds.count() << " milliseconds, "
                  << microseconds.count() << " microseconds, " 
                  << nanoseconds.count() << " nanoseconds" << std::endl;


        // Load updates (additions or deletions)
        int num_updates = 3000;            // Only one edge update in this batch
        cout<<"num_updates: "<<num_updates<<endl;
        // int update_edges[4] = {1, 8, 1, 6};   // Packed as: [source, destination]
        auto update_edges = gen_rand_edges(num_updates,nv);   // Packed as: [source, destination]
        bool isAddition = true;         // Indicate that this is an edge addition

       

        printf("Running BFS with edge additions...\n");
         start = std::chrono::high_resolution_clock::now();
        updateBFSUpdates(&custing2, levels, d_frontier, update_edges, num_updates, isAddition, 1);
        end = std::chrono::high_resolution_clock::now();
        
        // Calculate total duration
        duration =  std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
        
         hours = std::chrono::duration_cast<std::chrono::hours>(duration);
       minutes = std::chrono::duration_cast<std::chrono::minutes>(duration % std::chrono::hours(1));
      seconds = std::chrono::duration_cast<std::chrono::seconds>(duration % std::chrono::minutes(1));
         milliseconds = std::chrono::duration_cast<std::chrono::milliseconds>(duration % std::chrono::seconds(1));
        microseconds = std::chrono::duration_cast<std::chrono::microseconds>(duration % std::chrono::milliseconds(1));
        nanoseconds = std::chrono::duration_cast<std::chrono::nanoseconds>(duration % std::chrono::microseconds(1));

        std::cout << "Time taken for addition: "
                  << hours.count() << " hours, "
                  << minutes.count() << " minutes, "
                  << seconds.count() << " seconds, "
                  << milliseconds.count() << " milliseconds, "
                  << microseconds.count() << " microseconds, " 
                  << nanoseconds.count() << " nanoseconds" << std::endl;

        /////////////////////////
        //DELETION OF EDGE
        /////////////////////////
        int numEdgesToDelete = 2;
        BatchUpdateData bud(numEdgesToDelete, true, nv);  // Make sure nv is set to the number of vertices in your graph
        vertexId_t* src = bud.getSrc();
        vertexId_t* dst = bud.getDst();



// For bidirected edge deletion, 
{
  int update_edges_fwd1[] = {2, 4, 6, 7}; // first direction
  int update_edges_fwd2[] = {2, 4, 1, 8}; // first direction

  length_t numEdges = 2;  // One deletion update
  BatchUpdateData budFwd(numEdges, true);
  generateEdgeUpdates(nv, numEdges, budFwd.getSrc(), budFwd.getDst(), update_edges_fwd2);
  BatchUpdate buFwd(budFwd);
  custing2.edgeDeletions(buFwd);
  // Optionally verify deletion: 
  custing2.verifyEdgeDeletions(buFwd);
}

// {
//   int update_edges_rev1[] = { 4,2,7,6 }; // reverse direction
//   int update_edges_rev2[] = { 4,2,8,1 }; // reverse direction

//   length_t numEdges = 2;  // One deletion update
//   BatchUpdateData budRev(numEdges, true);
//   generateEdgeUpdates(nv, numEdges, budRev.getSrc(), budRev.getDst(), update_edges_rev2);
//   BatchUpdate buRev(budRev);
//   custing2.edgeDeletions(buRev);
//   // Optionally verify deletion: 
//   custing2.verifyEdgeDeletions(buRev);
// }
        int update_edges_del1[] = {2, 4, 6, 7}; // Edge 1->2 and 6->7
        // int update_edges_del2[] = {2, 4, 1, 8}; // Edge 1->2 and 6->7
        auto update_edges_del2 =gen_rand_edges(num_updates,nv);   // Edge 1->2 and 6->7


        int update_edges_batch_del1[] = {4,2,7,6}; // Edge 1->2 and 6->7
        int update_edges_batch_del2[] = {4,2,8,1}; // Edge 1->2 and 6->7


  
        printf("Running BFS for edge deletion...\n\n");

        start = std::chrono::high_resolution_clock::now();
        updateBFSUpdates(&custing2, levels, d_frontier, update_edges ,num_updates, false,1);
        end = std::chrono::high_resolution_clock::now();
        
        // Calculate total duration
        duration =  std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
        
         hours = std::chrono::duration_cast<std::chrono::hours>(duration);
       minutes = std::chrono::duration_cast<std::chrono::minutes>(duration % std::chrono::hours(1));
      seconds = std::chrono::duration_cast<std::chrono::seconds>(duration % std::chrono::minutes(1));
         milliseconds = std::chrono::duration_cast<std::chrono::milliseconds>(duration % std::chrono::seconds(1));
        microseconds = std::chrono::duration_cast<std::chrono::microseconds>(duration % std::chrono::milliseconds(1));
        nanoseconds = std::chrono::duration_cast<std::chrono::nanoseconds>(duration % std::chrono::microseconds(1));

        std::cout << "Time taken for deletion: "
                  << hours.count() << " hours, "
                  << minutes.count() << " minutes, "
                  << seconds.count() << " seconds, "
                  << milliseconds.count() << " milliseconds, "
                  << microseconds.count() << " microseconds, " 
                  << nanoseconds.count() << " nanoseconds" << std::endl;

        printf("Finished BFS for edge deletion.\n\n");
        custing2.freecuStinger();

    
    
    free(off);
    free(adj);
    return 0;	
}
