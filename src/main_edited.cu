#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>
#include <math.h>
#include <iostream>
#include <string>
#include <vector>
#include "utils.hpp"
#include "update.hpp"
#include "memoryManager.hpp"
#include "cuStinger.hpp"
#include <chrono>
#include <iomanip>

using namespace std;

// CUDA error checking macro
#define CHECK_CUDA(call) {                                            \
    hipError_t err = call;                                           \
    if (err != hipSuccess) {                                         \
        fprintf(stderr, "CUDA error at %s:%d: %s\n",                  \
                __FILE__, __LINE__, hipGetErrorString(err));         \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
}
struct Update {
    bool is_addition; // true if edge is added, false if removed
    vertexId_t u, v;  // edge between vertices u and v
};

//////////////////////////////////////
// BFS Kernel
//////////////////////////////////////
__global__ void bfs_kernel(const cuStinger::cusVertexData* dVD,
    const vertexId_t nv,
    const int current_level,
    const int* frontier,
    const int frontier_size,
    int* levels,
    vertexId_t* next_frontier,
    int* next_count) {
  // thread id
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < frontier_size) {
    int v = frontier[tid];  // current vertex from frontier
    int numNeighbors = dVD->used[v];
    // reinterpret pointer to edge memory
    vertexId_t* nbrArray = reinterpret_cast<vertexId_t*>(dVD->edMem[v]);
    for (int i = 0; i < numNeighbors; i++) {
      vertexId_t nbr = nbrArray[i];
      // If not yet discovered then set level and add to next frontier.
      if (atomicCAS(&levels[nbr], -1, current_level + 1) == -1) {
        int pos = atomicAdd(next_count, 1);
        next_frontier[pos] = nbr;
      }
    }
  }
}

//////////////////////////////////////
// Host-Side BFS Implementation
//////////////////////////////////////
int* runBFS(cuStinger* graph, int source ,int* h_levels,int* d_frontier) {
    // nv = number of vertices
    int nv = graph->nv;

    // array of levels on GPU
    int* d_levels;

    // if CUDA has error -> inform
    CHECK_CUDA(hipMalloc((void**)&d_levels, nv * sizeof(int)));

    // copy memory from host to device
    CHECK_CUDA(hipMemcpy(d_levels, h_levels, nv * sizeof(int), hipMemcpyHostToDevice));

    // next border
    int* d_next_frontier;
    // CHECK_CUDA(hipMalloc((void**)&d_frontier, nv * sizeof(vertexId_t)));
    CHECK_CUDA(hipMalloc((void**)&d_next_frontier, nv * sizeof(vertexId_t)));

    // Start frontier contains only the source vertex
    // how many elements in CPU queue
    int h_frontier_size = 1;
    CHECK_CUDA(hipMemcpy(d_frontier, &source, sizeof(int), hipMemcpyHostToDevice));

    // Allocate device counter for the next frontier
    int* d_next_count;
    CHECK_CUDA(hipMalloc((void**)&d_next_count, sizeof(int)));

    int current_level = 0;
    while (h_frontier_size > 0) {
        // Reset next frontier count
        CHECK_CUDA(hipMemset(d_next_count, 0, sizeof(int)));

        // Launch BFS kernel
        int blockSize = 256;
        // which thread runs where 
        int gridSize = (h_frontier_size + blockSize - 1) / blockSize;
        bfs_kernel<<<gridSize, blockSize>>>(graph->dVD, nv, current_level, d_frontier,
                                            h_frontier_size, d_levels, d_next_frontier, d_next_count);
        // cout<<"BFS KERNEL LAUNCHED"<<endl;
        CHECK_CUDA(hipDeviceSynchronize());

        // Copy next frontier size back to host
        int h_next_count;
        CHECK_CUDA(hipMemcpy(&h_next_count, d_next_count, sizeof(int), hipMemcpyDeviceToHost));

        // Swap frontiers
        int* temp = d_frontier;
        d_frontier = d_next_frontier;
        d_next_frontier = temp;

        h_frontier_size = h_next_count;
        current_level++;
    }

    // Copy levels array back to host
    CHECK_CUDA(hipMemcpy(h_levels, d_levels, nv * sizeof(int), hipMemcpyDeviceToHost));

    // Print BFS levels
    for (int i = 0; i < nv; i++) {
        printf("Vertex %d: Level %d\n", i, h_levels[i]);
    }

    // Free allocated memory
    // free(h_levels);
    CHECK_CUDA(hipFree(d_levels));
    CHECK_CUDA(hipFree(d_frontier));
    CHECK_CUDA(hipFree(d_next_frontier));
    CHECK_CUDA(hipFree(d_next_count));
    return h_levels;
}


//////////////////////////////////////
// BFS Update Kernel for Edge Additions
//////////////////////////////////////
// For each new edge (u,v), if u is discovered and can provide a shorter
// path to v, update v's level and add it to the update frontier.
__global__ void bfs_update_add_kernel(const cuStinger::cusVertexData* dVD,  
    const int* add_edges,   // packed as: [u0, v0, u1, v1, ...]  
    const int num_edges,  
    int* levels,  
    vertexId_t* update_frontier,  
    int* update_count) {  
  int tid = blockIdx.x * blockDim.x + threadIdx.x; 
  if (tid < num_edges) {  
  printf("tid: %d\n", tid); 

    int u = add_edges[2 * tid];  
    int v = add_edges[2 * tid + 1];  
    // Process only if u is discovered.
    printf("u: %d, v: %d\n", u, v);
    printf("fine");
    printf("levels[u]: %d\n", levels[u]);
    if (levels[u] != -1) {  
      int new_level = levels[u] + 1;  
      // Use atomicCAS to update levels[v] only if it is still undiscovered (-1)
      int old = atomicCAS(&levels[v], -1, new_level);  
      printf("old: %d\n", old);
      
      int old_2 = atomicMin(&levels[v], new_level);
      

      if (old == -1 || old_2>new_level) {  // Successful update: v was undiscovered  
        printf("update_count: %d\n", *(update_count));
        int pos = atomicAdd(update_count, 1);  
        printf("update_count: %d\n", *(update_count));
        printf("pos: %d\n", pos);   
        update_frontier[pos] = v;  
        printf("-------- chutiya code ------------ update_frontier[pos]: %d\n", update_frontier[pos]);
      }  
    }  
  }  
}

//////////////////////////////////////
// BFS Update Kernel for Edge Deletions
//////////////////////////////////////
// For each deleted edge (u,v), if v's current level comes from u then
// mark v for recomputation by setting its level to INT_MAX (as infinity)
// and adding it to the update frontier.
__global__ void bfs_update_del_kernel(const cuStinger::cusVertexData* dVD,
    const int* del_edges,   // packed as: [u0, v0, u1, v1, ...]
    const int num_edges,
    int* levels,
    vertexId_t* update_frontier,
    int* update_count) {
    
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num_edges) {
    int u = del_edges[2 * tid];
    int v = del_edges[2 * tid + 1];
    // Only process if (u,v) was used in the BFS tree:
    // u must have been reached and v’s level must be exactly levels[u] + 1.
    if (levels[u] == -1 || levels[v] != levels[u] + 1)
      return;
      
    // Mark v for recomputation: use atomicExch so that only one thread marks v.
    int old_level = atomicExch(&levels[v], INT_MAX);
    // If v was not already marked, add it to the update frontier.
    if(old_level != INT_MAX) {
      int pos = atomicAdd(update_count, 1);
      update_frontier[pos] = v;
    }
  }
}

//////////////////////////////////////
// Kernel to Invalidate Neighbor Levels
//////////////////////////////////////
// Marks neighbors of the invalidated nodes for recomputation.
__global__ void bfs_invalidate_levels_kernel(const cuStinger::cusVertexData* dVD,
    const int frontier_size,
    const vertexId_t* frontier,
    int* levels) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < frontier_size) {
    int v = frontier[tid];
    if (levels[v] == -1) {  // Process only invalidated nodes
      int numNeighbors = dVD->used[v];
      vertexId_t* nbrArray = reinterpret_cast<vertexId_t*>(dVD->edMem[v]);
      for (int i = 0; i < numNeighbors; i++) {
        int nbr = nbrArray[i];
        if (levels[nbr] > levels[v]) {  // Mark higher-level nodes for recomputation
          levels[nbr] = -1;
        }
      }
    }
  }
}

//////////////////////////////////////
// BFS Recompute Kernel for Propagation
//////////////////////////////////////
// This kernel is launched on the update frontier. For each vertex v in the
// frontier (which was marked with INT_MAX), it computes a new level by scanning
// all its neighbors and taking the minimum (levels[nbr] + 1). If a valid candidate
// is found, the new level is set (or -1 if no valid neighbor exists). Additionally,
// if v’s level changes, v’s children may need to update their levels so they are
// added to the next frontier.
__global__ void bfs_recompute_kernel(const cuStinger::cusVertexData* dVD,
    const int nv,
    const int* frontier,
    const int frontier_size,
    int* levels,
    vertexId_t* next_frontier,
    int* next_count) {
    
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < frontier_size) {
    int v = frontier[tid];
    // Recompute the new level for v.
    int candidate = INT_MAX;
    int numNeighbors = dVD->used[v];
    vertexId_t* nbrArray = reinterpret_cast<vertexId_t*>(dVD->edMem[v]);
    
    // For each neighbor, consider candidate level = levels[nbr] + 1.
    // Only consider neighbors with a valid (non-INT_MAX, non -1) level.
    for (int i = 0; i < numNeighbors; i++) {
      int nbr = nbrArray[i];
      int nbr_level = levels[nbr];
      if(nbr_level >= 0 && nbr_level != INT_MAX) {
        int old = min(candidate, nbr_level + 1);
      }
    }
    
    // If no valid candidate was found, v remains unreachable.
    int new_level = (candidate == INT_MAX) ? -1 : candidate;
    
    // Update v’s level if it differs from the (marked) value.
    // Note: v was marked as INT_MAX, so we expect new_level != INT_MAX.
    int old_level = INT_MAX;
    old_level = atomicExch(&levels[v], new_level);
    
    // If v's level changed (i.e. new_level is different from the old tree level),
    // then v's neighbors (which might be using v as a parent) could be affected.
    // We add v’s neighbors to the next frontier if they can improve their level.
    if (new_level != old_level) {
      for (int i = 0; i < numNeighbors; i++) {
        int nbr = nbrArray[i];
        // If neighbor has a valid level and its current level is greater than new_level + 1,
        // try to update it.
        if (levels[nbr] != -1 && levels[nbr] > new_level + 1) {
          int prev = atomicMin(&levels[nbr], new_level + 1);
          if (prev > new_level + 1) {
            int pos = atomicAdd(next_count, 1);
            next_frontier[pos] = nbr;
          }
        }
      }
    }
  }
}
//////////////////////////////////////
// BFS Kernel to handke updates
//////////////////////////////////////
__global__ void bfs_kernel_update(const cuStinger::cusVertexData* dVD,
    const vertexId_t nv,
    // int current_level,
    const int* frontier,
    const int frontier_size,
    int* levels,
    vertexId_t* next_frontier,
    int* next_count) {
  // thread id
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < frontier_size) {
    int v = frontier[tid];  // current vertex from frontier
    printf("----tid---: %d\n", tid);
    printf("---v---: %d\n", v);
    int numNeighbors = dVD->used[v];
    printf("----numNeighbors---: %d\n", numNeighbors);
    // reinterpret pointer to edge memory
    vertexId_t* nbrArray = reinterpret_cast<vertexId_t*>(dVD->edMem[v]);
    for (int i = 0; i < numNeighbors; i++) {
      vertexId_t nbr = nbrArray[i];
      printf("----nbr---: %d\n", nbr);
      int current_level = levels[v];
      printf("----current_level_: %d\n", current_level);
      // If not yet discovered then set level and add to next frontier.
      int old = atomicMin(&levels[nbr], current_level + 1);
      if (old == -1 || old > current_level + 1) {
        printf("Sauda chala\n");
        int pos = atomicAdd(next_count, 1);
        next_frontier[pos] = nbr;
      }
    }
  }
}

//////////////////////////////////////
// Host-Side Streaming Update Handler
//////////////////////////////////////
// This function applies a batch of update edges (additions or deletions)
// and then “propagates” the changes via BFS until no further level changes occur.
void updateBFSUpdates(cuStinger* graph, int* h_levels, int* d_frontier,
                      int* h_update_edges, int num_updates, bool isAddition) {
  int nv = graph->nv;

  // Copy update edges to device memory.
  int* d_update_edges;
  CHECK_CUDA(hipMalloc((void**)&d_update_edges, 2 * num_updates * sizeof(int)));
  CHECK_CUDA(hipMemcpy(d_update_edges, h_update_edges,
                        2 * num_updates * sizeof(int), hipMemcpyHostToDevice));

  int* d_levels;
  CHECK_CUDA(hipMalloc((void**)&d_levels, nv * sizeof(int)));

  CHECK_CUDA(hipMemcpy(d_levels, h_levels,
                        nv * sizeof(int), hipMemcpyHostToDevice));

  // Allocate temporary frontier and counter for propagating update changes.
  vertexId_t* d_update_frontier;
  vertexId_t* d_next_frontier;

  vertexId_t* h_update_frontier = (vertexId_t*)malloc(nv * sizeof(vertexId_t));
  CHECK_CUDA(hipMalloc((void**)&d_update_frontier, nv * sizeof(vertexId_t)));
  CHECK_CUDA(hipMalloc((void**)&d_next_frontier, nv * sizeof(vertexId_t)));

  int* d_update_count;
  CHECK_CUDA(hipMalloc((void**)&d_update_count, sizeof(int)));

  int h_update_count = 0;
  int blockSize = 256;
  int gridSize = (num_updates + blockSize - 1) / blockSize;

  // Launch the appropriate update kernel.
  if (isAddition) {
    bfs_update_add_kernel<<<gridSize, blockSize>>>(graph->dVD, d_update_edges, num_updates,
                                                     d_levels, d_update_frontier, d_update_count);

      // Get the count of vertices in the update frontier.
    CHECK_CUDA(hipMemcpy(&h_update_count, d_update_count, sizeof(int), hipMemcpyDeviceToHost));

  // Propagate the update changes until the frontier is empty.
    // int current_level = 0;  // You might want to adjust this if levels are relative to a root.
    printf("h_update_count: %d\n", h_update_count);
    while (h_update_count > 0) {

        // Reset the counter for the next propagation step.
        CHECK_CUDA(hipMemset(d_update_count, 0, sizeof(int)));
        gridSize = (h_update_count + blockSize - 1) / blockSize;
        // Use the original BFS kernel to propagate updated levels.
        bfs_kernel_update<<<gridSize, blockSize>>>(graph->dVD, nv, d_update_frontier,
                                            h_update_count, d_levels, d_next_frontier, d_update_count);
        CHECK_CUDA(hipDeviceSynchronize());
        // Swap frontiers.
        vertexId_t* temp;
        CHECK_CUDA(hipMalloc((void**)&temp, nv * sizeof(vertexId_t)));
        d_update_frontier = d_next_frontier;
        d_next_frontier = temp;
        // Get the new update frontier count.
        CHECK_CUDA(hipMemcpy(&h_update_count, d_update_count, sizeof(int), hipMemcpyDeviceToHost));
        printf("h_update_count: %d\n", h_update_count);

        // current_level++;
  }                                                 
   
  } else {
    bfs_update_del_kernel<<<gridSize, blockSize>>>(graph->dVD, d_update_edges, num_updates,
        d_levels, d_update_frontier, d_update_count);
CHECK_CUDA(hipDeviceSynchronize());
CHECK_CUDA(hipMemcpy(&h_update_count, d_update_count, sizeof(int), hipMemcpyDeviceToHost));

  // Propagate the update changes until the frontier is empty.
    // int current_level = 0;  // You might want to adjust this if levels are relative to a root.
    printf("h_update_count: %d\n", h_update_count);
    while (h_update_count > 0) {

        // Reset the counter for the next propagation step.
        CHECK_CUDA(hipMemset(d_update_count, 0, sizeof(int)));
        gridSize = (h_update_count + blockSize - 1) / blockSize;
        // Use the original BFS kernel to propagate updated levels.
       // For deletions, use the recompute kernel to recalc affected levels.
      bfs_recompute_kernel<<<gridSize, blockSize>>>(graph->dVD, nv, d_update_frontier,
        h_update_count, d_levels, d_next_frontier, d_update_count);

        CHECK_CUDA(hipDeviceSynchronize());
        // Swap frontiers.
        vertexId_t* temp;
        CHECK_CUDA(hipMalloc((void**)&temp, nv * sizeof(vertexId_t)));
        d_update_frontier = d_next_frontier;
        d_next_frontier = temp;
        // Get the new update frontier count.
        CHECK_CUDA(hipMemcpy(&h_update_count, d_update_count, sizeof(int), hipMemcpyDeviceToHost));
        printf("h_update_count: %d\n", h_update_count);

        // current_level++;
  }                       
}
  CHECK_CUDA(hipDeviceSynchronize());


   // Copy levels array back to host
   CHECK_CUDA(hipMemcpy(h_levels, d_levels, nv * sizeof(int), hipMemcpyDeviceToHost));
   // Print BFS levels
   for (int i = 0; i < nv; i++) {
    printf("Vertex %d: Level %d\n", i, h_levels[i]);
}

  // Free temporary device memory.
  CHECK_CUDA(hipFree(d_update_edges));
  CHECK_CUDA(hipFree(d_update_frontier));
  CHECK_CUDA(hipFree(d_update_count));
}


// Printer utility function for cuStinger
void printcuStingerUtility(cuStinger custing, bool allInfo) {
    length_t used, allocated;
    used = custing.getNumberEdgesUsed();
    allocated = custing.getNumberEdgesAllocated();
    if (allInfo)
        cout << "," << used << "," << allocated << "," << (float)used / (float)allocated;  
    else
        cout << "," << (float)used / (float)allocated;
}

// Generate random edge updates
void generateEdgeUpdates(length_t nv, length_t numEdges, vertexId_t* edgeSrc, vertexId_t* edgeDst) {
    for (int e = 0; e < numEdges; e++) {
        edgeSrc[e] = rand() % nv;
        edgeDst[e] = rand() % nv;
    }
}

// RMAT edge generation (helper functions)
typedef struct dxor128_env {
    unsigned x, y, z, w;
} dxor128_env_t;

double dxor128(dxor128_env_t * e) {
    unsigned t = e->x ^ (e->x << 11);
    e->x = e->y; e->y = e->z; e->z = e->w;
    e->w = (e->w ^ (e->w >> 19)) ^ (t ^ (t >> 8));
    return e->w * (1.0 / 4294967296.0);
}

void dxor128_init(dxor128_env_t * e) {
    e->x = 123456789;
    e->y = 362436069;
    e->z = 521288629;
    e->w = 88675123;
}

void dxor128_seed(dxor128_env_t * e, unsigned seed) {
    e->x = 123456789;
    e->y = 362436069;
    e->z = 521288629;
    e->w = seed;
}

void rmat_edge(int64_t * iout, int64_t * jout, int SCALE, double A, double B, double C, double D, dxor128_env_t * env)
{
    int64_t i = 0, j = 0;
    int64_t bit = ((int64_t) 1) << (SCALE - 1);

    while (1) {
        const double r = ((double) rand() / (RAND_MAX));
        if (r > A) {
            if (r <= A + B)
                j |= bit;
            else if (r <= A + B + C)
                i |= bit;
            else {
                j |= bit;
                i |= bit;
            }
        }
        if (1 == bit)
            break;

        A *= (9.5 + ((double) rand() / (RAND_MAX))) / 10;
        B *= (9.5 + ((double) rand() / (RAND_MAX))) / 10;
        C *= (9.5 + ((double) rand() / (RAND_MAX))) / 10;
        D *= (9.5 + ((double) rand() / (RAND_MAX))) / 10;

        {
            const double norm = 1.0 / (A + B + C + D);
            A *= norm; B *= norm; C *= norm;
        }
        D = 1.0 - (A + B + C);
        bit >>= 1;
    }
    *iout = i;
    *jout = j;
}

// Generate RMAT edge updates
void generateEdgeUpdatesRMAT(length_t nv, length_t numEdges, vertexId_t* edgeSrc, vertexId_t* edgeDst,
                             double A, double B, double C, double D, dxor128_env_t * env)
{
    int64_t src, dst;
    int scale = (int)log2(double(nv));
    for (int32_t e = 0; e < numEdges; e++) {
        rmat_edge(&src, &dst, scale, A, B, C, D, env);
        edgeSrc[e] = src;
        edgeDst[e] = dst;
    }
}

int main(const int argc, char *argv[])
{  
    //  testBFS();
    int device = 0;
    hipSetDevice(device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
 
    length_t nv, ne, *off;
    vertexId_t *adj;
    int isRmat = 0;

    if (argc < 3) {
        cout << "Usage: " << argv[0] << " <graph_file> <graphName> [options]\n";
        return 1;
    }
 
    char* graphName = argv[2];
    srand(100);

    bool isDimacs, isSNAP, isMM;
    string filename(argv[1]);
    isDimacs = (filename.find(".graph") != string::npos);
    isSNAP   = (filename.find(".txt") != string::npos);
    isMM     = (filename.find(".mtx") != string::npos);
    isRmat   = (filename.find("kron") != string::npos);

    bool undirected = hasOption("--undirected", argc, argv);

    if (isDimacs) {
        readGraphDIMACS(argv[1], &off, &adj, &nv, &ne, isRmat);
    } else if (isSNAP) {
        readGraphSNAP(argv[1], &off, &adj, &nv, &ne, undirected);
    } else if (isMM) {
        readGraphMatrixMarket(argv[1], &off, &adj, &nv, &ne, undirected);
    } else { 
        cout << "Unknown graph type" << endl;
        return 1;
    }
 

    // Set up for cuStinger
    hipEvent_t ce_start, ce_stop;
    cuStingerInitConfig hipInit;
    hipInit.initState   = eInitStateCSR;
    hipInit.maxNV       = nv + 1;
    hipInit.useVWeight  = false;
    hipInit.isSemantic  = false;
    hipInit.useEWeight  = false;
    
    hipInit.csrNV       = nv;
    hipInit.csrNE       = ne;
    hipInit.csrOff      = off;
    hipInit.csrAdj      = adj;
    hipInit.csrVW       = NULL;
    hipInit.csrEW       = NULL;
    cuStinger custing2(defaultInitAllocater, defaultUpdateAllocater);
    custing2.initializeCuStinger(hipInit);
    // Run BFS on the original CSR arrays
    cout << "Running BFS on the input graph ...\n";
    int* h_levels = (int*)malloc(nv * sizeof(int));
    for (int i = 0; i < nv; i++) {
        h_levels[i] = -1;  // -1 indicates undiscovered
    }
    h_levels[1] = 0;
    int* d_frontier;
    CHECK_CUDA(hipMalloc((void**)&d_frontier, nv * sizeof(vertexId_t)));
    auto start = std::chrono::high_resolution_clock::now();
    int * levels = runBFS(&custing2, 1 ,h_levels,d_frontier);
    auto end = std::chrono::high_resolution_clock::now();

    cout << "Finished BFS.\n\n";
    cout<<"nv: "<<hipInit.csrNV<<" "<<"ne: "<<hipInit.csrNE<<endl;  

    // Calculate the duration
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);

    // Convert duration to hours, minutes, seconds, and milliseconds
    auto hours = std::chrono::duration_cast<std::chrono::hours>(duration);
    duration -= hours;
    auto minutes = std::chrono::duration_cast<std::chrono::minutes>(duration);
    duration -= minutes;
    auto seconds = std::chrono::duration_cast<std::chrono::seconds>(duration);
    duration -= seconds;
    auto milliseconds = std::chrono::duration_cast<std::chrono::milliseconds>(duration);

    // Display the elapsed time
    std::cout << "Time taken for BFS: "
              << hours.count() << " hours, "
              << minutes.count() << " minutes, "
              << seconds.count() << " seconds, "
              << milliseconds.count() << " milliseconds" << std::endl;


    // 
    ////////////////
    //TESTING STREAMING BFS
    ////////////////////
        // Load updates (additions or deletions)
        int update_edges[2] = {1, 6};   // Packed as: [source, destination]
        int num_updates = 1;            // Only one edge update in this batch
        bool isAddition = true;         // Indicate that this is an edge addition


        updateBFSUpdates(&custing2, levels, d_frontier, update_edges,num_updates, isAddition);

    free(off);
    free(adj);
    return 0;	
}
